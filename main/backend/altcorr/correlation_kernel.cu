#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <THC/THCAtomics.cuh>
#include <vector>
#include <iostream>

using namespace torch::indexing;

#define THREADS 256
#define BLOCKS(n) (n + THREADS - 1) / THREADS

__forceinline__ __device__
bool within_bounds(int h, int w, int H, int W) {
  return h >= 0 && h < H && w >= 0 && w < W;
}

template <typename scalar_t>
__global__ void patchify_forward_kernel(int R,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> net,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> coords,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> patches)
{
  // diameter
  const int D = 2*R + 2;

  const int B = coords.size(0);
  const int M = coords.size(1);
  const int C = net.size(1);
  const int H = net.size(2);
  const int W = net.size(3);

  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < B * M * D * D) {
    const int ii = n % D; n /= D;
    const int jj = n % D; n /= D;
    const int  m = n % M; n /= M;

    const float x = coords[n][m][0];
    const float y = coords[n][m][1];
    const int i = static_cast<int>(floor(y)) + (ii - R);
    const int j = static_cast<int>(floor(x)) + (jj - R);

    if (within_bounds(i, j, H, W)) {
      for (int k=0; k<C; k++)
        patches[n][m][k][ii][jj] = net[n][k][i][j];
    }
  }
}

template <typename scalar_t>
__global__ void patchify_backward_kernel(int R,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> patch_gradient,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> coords,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> gradient)
{
  // diameter
  const int D = 2*R + 2;

  const int B = coords.size(0);
  const int M = coords.size(1);
  const int C = gradient.size(1);
  const int H = gradient.size(2);
  const int W = gradient.size(3);

  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < B * M * D * D) {
    const int ii = n % D; n /= D;
    const int jj = n % D; n /= D;
    const int  m = n % M; n /= M;

    const float x = coords[n][m][0];
    const float y = coords[n][m][1];
    const int i = static_cast<int>(floor(y)) + (ii - R);
    const int j = static_cast<int>(floor(x)) + (jj - R);

    if (within_bounds(i, j, H, W)) {
      for (int k=0; k<C; k++)
        atomicAdd(&gradient[n][k][i][j], patch_gradient[n][m][k][ii][jj]);
    }
  }
}

template <typename scalar_t>
__global__ void corr_forward_kernel(int R,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap1,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap2,
    const torch::PackedTensorAccessor32<float,5,torch::RestrictPtrTraits> coords,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> us,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> vs,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> corr)
{
  // diameter
  const int D = 2*R + 2;

  const int B = coords.size(0);
  const int M = coords.size(1);
  const int H = coords.size(3);
  const int W = coords.size(4);

  const int C = fmap1.size(2);
  const int H2 = fmap2.size(3);
  const int W2 = fmap2.size(4);

  int n = blockIdx.x * blockDim.x + threadIdx.x;

  if (n < B * M * H * W * D * D) {
    const int jj = n % D; n /= D;
    const int ii = n % D; n /= D;
    const int j0 = n % W; n /= W;
    const int i0 = n % H; n /= H;
    const int  m = n % M; n /= M;

    const int ix = us[m];
    const int jx = vs[m];

    const float x = coords[n][m][0][i0][j0];
    const float y = coords[n][m][1][i0][j0];

    const int i1 = static_cast<int>(floor(y)) + (ii - R);
    const int j1 = static_cast<int>(floor(x)) + (jj - R);

    scalar_t s = 0;
    if (within_bounds(i1, j1, H2, W2)) {

      #pragma unroll 8
      for (int i=0; i<C; i+=8) {
        scalar_t f1[8]; for (int j=0; j<8; j++) f1[j] = fmap1[n][ix][i+j][i0][j0];
        scalar_t f2[8]; for (int j=0; j<8; j++) f2[j] = fmap2[n][jx][i+j][i1][j1];

        #pragma unroll
        for (int j=0; j<8; j++) s += f1[j] * f2[j];
      }
    }

    corr[n][m][ii][jj][i0][j0] = s;
  }
}


template <typename scalar_t>
__global__ void corr_backward_kernel(int R,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap1,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap2,
    const torch::PackedTensorAccessor32<float,5,torch::RestrictPtrTraits> coords,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> us,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> vs,
    const torch::PackedTensorAccessor32<float,6,torch::RestrictPtrTraits> corr_grad,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap1_grad,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap2_grad)
{
  // diameter
  const int D = 2*R + 2;

  const int B = coords.size(0);
  const int M = coords.size(1);
  const int H = coords.size(3);
  const int W = coords.size(4);

  const int C = fmap1.size(2);
  const int H2 = fmap2.size(3);
  const int W2 = fmap2.size(4);

  int n = blockIdx.x * blockDim.x + threadIdx.x;

  if (n < B * M * H * W * D * D) {
    const int jj = n % D; n /= D;
    const int ii = n % D; n /= D;
    const int j0 = n % W; n /= W;
    const int i0 = n % H; n /= H;
    const int  m = n % M; n /= M;

    const int ix = us[m];
    const int jx = vs[m];

    const float x = coords[n][m][0][i0][j0];
    const float y = coords[n][m][1][i0][j0];

    const int i1 = static_cast<int>(floor(y)) + (ii - R);
    const int j1 = static_cast<int>(floor(x)) + (jj - R);

    const scalar_t g = (scalar_t) corr_grad[n][m][ii][jj][i0][j0];

    if (within_bounds(i1, j1, H2, W2)) {
      #pragma unroll 32
      for (int i=0; i<C; i++) {
        atomicAdd(&fmap1_grad[n][ix][i][i0][j0], g * fmap2[n][jx][i][i1][j1]);
        atomicAdd(&fmap2_grad[n][jx][i][i1][j1], g * fmap1[n][ix][i][i0][j0]);
      }
    }
  }
}


std::vector<torch::Tensor> corr_cuda_forward(
  torch::Tensor fmap1,
  torch::Tensor fmap2,
  torch::Tensor coords,
  torch::Tensor ii,
  torch::Tensor jj,
  int radius)
{
  const int B = coords.size(0);
  const int M = coords.size(1);

  const int H = coords.size(3);
  const int W = coords.size(4);
  const int D = 2 * radius + 2;

  auto opts = fmap1.options();
  auto corr = torch::empty({B, M, D, D, H, W}, opts);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(fmap1.type(), "corr_forward_kernel", ([&] {
      corr_forward_kernel<scalar_t><<<BLOCKS(B * M * H * W * D * D), THREADS>>>(radius,
        fmap1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        fmap2.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        coords.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
        ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
        jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
        corr.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>());
  }));

  torch::Tensor x = coords.index({Slice(), Slice(), 0, None, None});
  torch::Tensor y = coords.index({Slice(), Slice(), 1, None, None});
  torch::Tensor dx = x - x.floor(); dx = dx.to(fmap1.dtype());
  torch::Tensor dy = y - y.floor(); dy = dy.to(fmap2.dtype());

  torch::Tensor out;
  out  = (1 - dx) * (1 - dy) * corr.index({Slice(), Slice(), Slice(0, D-1), Slice(0, D-1)});
  out +=     (dx) * (1 - dy) * corr.index({Slice(), Slice(), Slice(0, D-1), Slice(1, D-0)});
  out += (1 - dx) *     (dy) * corr.index({Slice(), Slice(), Slice(1, D-0), Slice(0, D-1)});
  out +=     (dx) *     (dy) * corr.index({Slice(), Slice(), Slice(1, D-0), Slice(1, D-0)});

  return { out.permute({0,1,3,2,4,5}) };
}


std::vector<torch::Tensor> corr_cuda_backward(
  torch::Tensor fmap1,
  torch::Tensor fmap2,
  torch::Tensor coords,
  torch::Tensor ii,
  torch::Tensor jj,
  torch::Tensor grad,
  int radius)
{
  const int B = coords.size(0);
  const int M = coords.size(1);

  const int H = coords.size(3);
  const int W = coords.size(4);
  const int D = 2 * radius + 2;
   
  grad = grad.permute({0,1,3,2,4,5}).contiguous();
  torch::Tensor x = coords.index({Slice(), Slice(), 0, None, None});
  torch::Tensor y = coords.index({Slice(), Slice(), 1, None, None});
  torch::Tensor dx = x - x.floor();
  torch::Tensor dy = y - y.floor();

  auto opts = torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA);
  torch::Tensor g1 = torch::zeros({B, M, D, D, H, W}, grad.options());
  torch::Tensor g2 = torch::zeros({B, M, D, D, H, W}, grad.options());
  torch::Tensor g3 = torch::zeros({B, M, D, D, H, W}, grad.options());
  torch::Tensor g4 = torch::zeros({B, M, D, D, H, W}, grad.options());
  
  g1.index_put_({Slice(), Slice(), Slice(0, D-1), Slice(0, D-1)}, (1 - dx) * (1 - dy) * grad);
  g2.index_put_({Slice(), Slice(), Slice(0, D-1), Slice(1, D-0)},     (dx) * (1 - dy) * grad); 
  g3.index_put_({Slice(), Slice(), Slice(1, D-0), Slice(0, D-1)}, (1 - dx) *     (dy) * grad);
  g4.index_put_({Slice(), Slice(), Slice(1, D-0), Slice(1, D-0)},     (dx) *     (dy) * grad);

  torch::Tensor corr_grad = g1 + g2 + g3 + g4;
  auto fmap1_grad = torch::zeros_like(fmap1);
  auto fmap2_grad = torch::zeros_like(fmap2);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(fmap1.type(), "corr_backward_kernel", ([&] {
    corr_backward_kernel<scalar_t><<<BLOCKS(B * M * H * W * D * D), THREADS>>>(radius,
      fmap1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
      fmap2.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
      coords.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
      ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      corr_grad.packed_accessor32<float,6,torch::RestrictPtrTraits>(),
      fmap1_grad.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
      fmap2_grad.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>());
  }));

  return {fmap1_grad, fmap2_grad};
}

std::vector<torch::Tensor> patchify_cuda_forward(
  torch::Tensor net, torch::Tensor coords, int radius)
{
  const int B = coords.size(0);
  const int M = coords.size(1);
  const int C = net.size(1);
  const int D = 2 * radius + 2;

  auto opts = net.options();
  auto patches = torch::zeros({B, M, C, D, D}, opts);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(net.type(), "patchify_forward_kernel", ([&] {
      patchify_forward_kernel<scalar_t><<<BLOCKS(B * M * D * D), THREADS>>>(radius,
        net.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        coords.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        patches.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>());
  }));

  return { patches };
}


std::vector<torch::Tensor> patchify_cuda_backward(
  torch::Tensor net,
  torch::Tensor coords,
  torch::Tensor gradient,
  int radius)
{
  const int B = coords.size(0);
  const int M = coords.size(1);
  const int C = net.size(1);
  const int H = net.size(2);
  const int W = net.size(3);
  const int D = 2 * radius + 2;
  
  torch::Tensor net_gradient = torch::zeros_like(net);
  
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(net.type(), "patchify_backward_kernel", ([&] {
    patchify_backward_kernel<scalar_t><<<BLOCKS(B * M * D * D), THREADS>>>(radius,
      gradient.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
      coords.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      net_gradient.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>());
  }));

  return { net_gradient };
}